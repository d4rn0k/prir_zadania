#include "hip/hip_runtime.h"
#include "macierz_gpu.h"

#include <iostream>
#include <math.h>

int BLOCK_SIZE;

__global__ void matrixMultiplyKernel(Matrix A, Matrix B, Matrix C) {

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int matrixSize = A.matrixSize;

	float tempCValue = 0.0f;

	//	printf("row(%d) = threadIdx.y(%d) + blockIdx.y(%d) * blockDim.y(%d),  col(%d) = threadIdx.x(%d) + blockIdx.x(%d) * blockDim.x(%d)\n",
	//			row, threadIdx.y, blockIdx.y, blockDim.y, col, threadIdx.x, blockIdx.x, blockDim.x);

	if (row < matrixSize && col < matrixSize){

		for(int k = 0; k < matrixSize; k++) {
			tempCValue += GetElement(A, row, k) * GetElement(B, k, col);
		}

		//__syncthreads();
		SetElement(C, row, col, tempCValue);
	}
}

void matrixMultiplyAndGenerateHost(Matrix cpu_A, Matrix cpu_B, Matrix cpu_C) {

	float elapsedTime;
	Matrix dev_A, dev_B, dev_C;
	hipEvent_t startTime, stopTime;
	size_t totalSize = cpu_A.matrixSize * cpu_A.matrixSize * sizeof(float);

	//	printf("Alokujemy %lu B, %lu KB pamieci, na %lu  elementow\n", totalSize, totalSize / 1024, totalSize / sizeof(float) );

	// Deklarujemy  siatkę i wątki na blok
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	dim3 grid( (int) ceil(cpu_A.matrixSize/ (float)threadsPerBlock.x),(int) ceil(cpu_A.matrixSize / (float)threadsPerBlock.y));

	dev_A.matrixSize = dev_B.matrixSize = dev_C.matrixSize = cpu_A.matrixSize;

	//Alokacja macierzy dev_A, dev_B, dev_C
	CUDA_CHECK_RETURN( hipMalloc(&dev_A.elements, totalSize) );
	CUDA_CHECK_RETURN( hipMalloc(&dev_B.elements, totalSize) );
	CUDA_CHECK_RETURN( hipMalloc(&dev_C.elements, totalSize) );

	// Generowanie macierzy!
	generateMatrixes(dev_A, dev_B, dev_A.matrixSize);

	// Jeśli wymiar macierzy == liczba bloków
	if (grid.x == 1) {
		grid = dim3(1, 1);
	}

	CUDA_CHECK_RETURN( hipEventCreate ( &startTime));
	CUDA_CHECK_RETURN( hipEventCreate ( &stopTime));

	// start mierzenia czasu
	CUDA_CHECK_RETURN( hipEventRecord(startTime, 0) );


	//printf("Uruchamiamy kernel z parametrami: threadsPerBlock.x = %d, threadsPerBlock.y = %d, grid.x = %d, grid.y = %d\n", threadsPerBlock.x, threadsPerBlock.y, grid.x, grid.y);


	// Wywołanie kernela
	matrixMultiplyKernel<<< grid , threadsPerBlock >>> (dev_A, dev_B, dev_C);

	CUDA_CHECK_RETURN( hipEventRecord(stopTime, 0));
	CUDA_CHECK_RETURN( hipEventSynchronize(stopTime));
	CUDA_CHECK_RETURN( hipEventElapsedTime(&elapsedTime, startTime, stopTime));
	printf("Czas wykonania: %.0lf ms\n",elapsedTime);


	//	printf("Uruchamiamy kernel z parametrami: threadsPerBlock.x = %d, threadsPerBlock.y = %d, grid.x = %d, grid.y = %d\n",
	//			threadsPerBlock.x, threadsPerBlock.y, grid.x, grid.y);

	// Kopiowanie macierzy z GPU na CPU
	CUDA_CHECK_RETURN(hipMemcpy(cpu_C.elements, dev_C.elements, totalSize, hipMemcpyDeviceToHost));

	// Kopiowanie wygenerowanych macierzy A, B pomocnicze
	//	CUDA_CHECK_RETURN(hipMemcpy(cpu_A.elements, dev_A.elements, totalSize, hipMemcpyDeviceToHost));
	//	CUDA_CHECK_RETURN(hipMemcpy(cpu_B.elements, dev_B.elements, totalSize, hipMemcpyDeviceToHost));

	//  Sprawdza niektóre elementy macierzy wynikowej cpu_C do testu poprawności!
	//		checkMatrix(cpu_C);


	//Czyszczenie macierzy dev_A, dev_B, dev_C na urządzaniu
	hipFree(dev_A.elements);
	hipFree(dev_B.elements);
	hipFree(dev_C.elements);
}

// Pobiera element z macierzy
__device__ float GetElement(const Matrix A, int row, int col) {
	return A.elements[row * A.matrixSize + col];
}

// Ustawianie konkretnego elementu
__device__ void SetElement(Matrix A, int row, int col, float value) {
	//printf("Ustawiamy element macierzy[%d] = %f\n", row * A.stride + col, value);
	A.elements[row * A.matrixSize + col] = value;
}

__global__ void generateMatrixesKernel(Matrix dev_A, Matrix dev_B, int matrixSize){

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	if (row < matrixSize && col < matrixSize ){

		float AVal = fmod( (float) ( sinf( (float) row) * (row * col)), 10.0f );
		float BVal = fmod( (float) ( cosf( (float) col) * (row + col)), 10.0f );

		SetElement(dev_A, row, col, AVal);
		SetElement(dev_B, row, col, BVal);
	}

}


// Funkcja zlecająca wypełnianianie macierzy
void generateMatrixes(Matrix A, Matrix B, int matrixSize){

	dim3 threadsPerBlock(32, 32);
	dim3 grid(matrixSize/threadsPerBlock.x, matrixSize/threadsPerBlock.y);

	//kernel<<<blocks, threads>>> (int param1, float param2);
	generateMatrixesKernel<<<grid, threadsPerBlock>>> (A, B, matrixSize);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	//printf("Wygenerowano Macierze!\n");
}


int main(int argc, char* argv[]){

	Matrix cpu_A, cpu_B, cpu_C;

	int matrixSize;
	size_t totalSize;

	if (argc != 2 || ( (atoi(argv[1])) <= 0) ) {
		printf("Błąd, złe parametry\n\nWywołanie: %s <wymiar boku macierzy>\n\n", argv[0]);
		exit(-1);
	}

	//CUDA_CHECK_RETURN( hipSetDevice(1) );
	matrixSize = atoi(argv[1]);
	BLOCK_SIZE = 16;

	totalSize = matrixSize * matrixSize * sizeof(float);

	//	printf("Obliczamy macierze wymiarów: %d x %d\n", matrixSize, matrixSize);

	cpu_A.matrixSize = cpu_B.matrixSize = cpu_C.matrixSize = matrixSize;

	cpu_A.elements = (float*)malloc(totalSize);
	cpu_B.elements = (float*)malloc(totalSize);
	cpu_C.elements = (float*)malloc(totalSize);

	CUDA_CHECK_RETURN(hipSetDevice(1));
	matrixMultiplyAndGenerateHost(cpu_A, cpu_B, cpu_C);

	//
	//	Matrix cpu_B_after_gen;
	//	cpu_B_after_gen.matrixSize = matrixSize;
	//	cpu_B_after_gen.elements = (float*)malloc(totalSize);
	//	CUDA_CHECK_RETURN(hipMemcpy(cpu_B_after_gen.elements, cpu_B.elements, totalSize, hipMemcpyDeviceToHost));

	return 0;
}

void checkMatrix(Matrix cpu_A) {

	int indexes[] = { 0, 5, 10, cpu_A.matrixSize / 2, cpu_A.matrixSize - 1 };

	for (size_t i = 0; i < sizeof(indexes) / sizeof(int); i ++){
		int mIndex = indexes[i];
		printf("Wartość dev_C[%d][%d] = %f\n", mIndex, mIndex, cpu_A.elements[mIndex* cpu_A.matrixSize + mIndex]);
	}
}

void printMatrix(Matrix matrix, int rows, int cols){
	for(int i = 0; i < rows; i++){
		for(int j = 0; j < cols; j++)
			printf("%7.2f ", matrix.elements[i*matrix.matrixSize + j]);
		printf("\n");
	}
	printf("\n");
}

static void checkCudaError (const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess) {
		return;
	}
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
